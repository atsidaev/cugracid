#include "hip/hip_runtime.h"
#include <stdio.h>

#define SIDE 256

#include "float.c"
;
const int dsize = sizeof(float);

__device__
float Vz1(float x, float y, float xi, float nu, float z1, float z2, float H)
{
	float x_dif = (xi - x);
	float y_dif = (nu - y);
	float z_dif2 = (z2 - H);
	float z_dif1 = (z1 - H);

	float R1 = sqrt(x_dif * x_dif + y_dif * y_dif + z_dif1 * z_dif1);
	float R2 = sqrt(x_dif * x_dif + y_dif * y_dif + z_dif2 * z_dif2);

	return 
		(nu == y ? 0 : y_dif * log((x_dif + R2) / (x_dif + R1))) + 
		(xi == x ? 0 : x_dif * log((y_dif + R2) / (y_dif + R1))) -

		((z_dif2 == 0 ? 0 : z_dif2 * atan(x_dif * y_dif / (z_dif2 * R2))) -
		(z_dif1 == 0 ? 0 : z_dif1 * atan(x_dif * y_dif / (z_dif1 * R1))));
}

__device__
float Vz2(float x, float y, float xi, float y1, float y2, float z1, float z2, float H)
{
	return Vz1(x, y, xi, y2, z1, z2, H) - Vz1(x, y, xi, y1, z1, z2, H);
}

__device__
float Vz3(float x, float y, float x1, float x2, float y1, float y2, float z1, float z2, float H)
{
	return Vz2(x, y, x2, y1, y2, z1, z2, H) - Vz2(x, y, x1, y1, y2, z1, z2, H);
}

__global__
void Calculate(int yLine, float xLL, float yLL, float xStep, float yStep, float* top, float* bottom, float* result)
{
	float x = xLL + xStep * threadIdx.x;
	float y = yLL + yStep * yLine;
	
	float x1 = xLL + xStep * blockIdx.x;
	float x2 = x1 + xStep;
	
	float y1 = yLL + yStep * blockIdx.y;
	float y2 = y1 + yStep;
	
	int pos_grid = blockIdx.x + blockIdx.y * SIDE;
	float t = 1; //top[pos_grid];
	float b = 0; //bottom[pos_grid];
	
	int pos_result = threadIdx.x + yLine * SIDE;
	
	float r = Vz3(x, y, x1, x2, y1, y2, t, b, 0);
	
	//atomicAdd(&result[pos_result], r);
	result[pos_result] += r;
}

int main()
{
	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, 0);
	printf("%s API version %d.%d\n", props.name, props.major, props.minor);
        
	printf("Max block dimensions: [%d, %d, %d]\n", props.maxThreadsDim[0], props.maxThreadsDim[1], props.maxThreadsDim[2]);
	printf("Threads per block: %d\n", props.maxThreadsPerBlock);
	
	float *result = (float*)malloc(SIDE * SIDE * dsize);
	memset(result, 0, SIDE * SIDE * dsize);
	
	float *resultd, *catd, *zerod;
	hipMalloc((void**)&resultd, SIDE * SIDE * dsize);
	hipMalloc((void**)&catd, SIDE * SIDE * dsize);
	hipMalloc((void**)&zerod, SIDE * SIDE * dsize);
	
	hipMemcpy(zerod, result, SIDE * SIDE * dsize, hipMemcpyHostToDevice);
	hipMemcpy(resultd, result, SIDE * SIDE * dsize, hipMemcpyHostToDevice);
	hipMemcpy(catd, cat, SIDE * SIDE * dsize, hipMemcpyHostToDevice);
	
	dim3 blocks(256, 256);
	dim3 threads(256);
	
	//Vz3<<<blocks,threads>>>(0, 0, -100000000, 100000000, -100000000, 100000000, 1, 0, 0, result);
	//V<<<1,1>>>(result);
	Calculate<<<blocks,threads>>>(128, 10000, 11000, 4, 4, zerod, catd, resultd);
	
	hipError_t error = hipGetLastError();
	printf("Error: %s\n", hipGetErrorString(error));
	
	hipMemcpy(result, resultd, SIDE * SIDE * dsize, hipMemcpyDeviceToHost);
	hipFree( resultd );
	printf("%f\n", result[128 * 256 + 128]);
	
	return 0;
}
