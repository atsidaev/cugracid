#include "hip/hip_runtime.h"
#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

#include <math.h>

#define SIDE 256
#define FLOAT double
const int dsize = sizeof(FLOAT);

__device__
FLOAT Vz1(FLOAT x, FLOAT y, FLOAT xi, FLOAT nu, FLOAT z1, FLOAT z2, FLOAT H)
{
	FLOAT x_dif = (xi - x);
	FLOAT y_dif = (nu - y);
	FLOAT z_dif2 = (z2 - H);
	FLOAT z_dif1 = (z1 - H);

	FLOAT R1 = sqrt(x_dif * x_dif + y_dif * y_dif + z_dif1 * z_dif1);
	FLOAT R2 = sqrt(x_dif * x_dif + y_dif * y_dif + z_dif2 * z_dif2);

	return 
		(nu == y ? 0 : y_dif * log((x_dif + R2) / (x_dif + R1))) + 
		(xi == x ? 0 : x_dif * log((y_dif + R2) / (y_dif + R1))) -

		((z_dif2 == 0 ? 0 : z_dif2 * atan(x_dif * y_dif / (z_dif2 * R2))) -
		(z_dif1 == 0 ? 0 : z_dif1 * atan(x_dif * y_dif / (z_dif1 * R1))));
}

__device__
FLOAT Vz2(FLOAT x, FLOAT y, FLOAT xi, FLOAT y1, FLOAT y2, FLOAT z1, FLOAT z2, FLOAT H)
{
	return Vz1(x, y, xi, y2, z1, z2, H) - Vz1(x, y, xi, y1, z1, z2, H);
}

__device__
FLOAT Vz3(FLOAT x, FLOAT y, FLOAT x1, FLOAT x2, FLOAT y1, FLOAT y2, FLOAT z1, FLOAT z2, FLOAT H)
{
	return Vz2(x, y, x2, y1, y2, z1, z2, H) - Vz2(x, y, x1, y1, y2, z1, z2, H);
}

__global__
void Calculate(int yLine, FLOAT xLL, FLOAT yLL, FLOAT xStep, FLOAT yStep, FLOAT* top, FLOAT* bottom, FLOAT* result, FLOAT* sync)
{
	FLOAT x = xLL + xStep * blockIdx.x;
	FLOAT y = yLL + yStep * blockIdx.y;
	
	FLOAT x1 = xLL + xStep * threadIdx.x;
	FLOAT x2 = x1 + xStep;
	
	FLOAT y1 = yLL + yStep * yLine;
	FLOAT y2 = y1 + yStep;
	
	int pos_grid = threadIdx.x + yLine * SIDE;
	FLOAT t = 1; //top[pos_grid];
	FLOAT b = 0; //bottom[pos_grid];
	
	int pos_result = blockIdx.x + blockIdx.y * SIDE;
	
	FLOAT r = Vz3(x, y, x1, x2, y1, y2, t, b, 0);
	
	//atomicAdd(&result[pos_result], r);
	//result[pos_result] += r;
	sync[pos_result * SIDE + threadIdx.x] = r;
	FLOAT res = result[pos_result];
	__syncthreads();
	for (int i = 0, p = pos_result * SIDE; i < SIDE; i++, p++)
		res += sync[p];
	result[pos_result] = res;
}

int main()
{
	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, 0);
	printf("%s API version %d.%d\n", props.name, props.major, props.minor);
	printf("Maximum texture dimensions: 1D: %d, 2D: %d, 3D: %d\n", props.maxTexture1D, props.maxTexture2D, props.maxTexture3D);
    
	printf("Max block dimensions: [%d, %d, %d]\n", props.maxThreadsDim[0], props.maxThreadsDim[1], props.maxThreadsDim[2]);
	printf("Threads per block: %d\n", props.maxThreadsPerBlock);
	
	FLOAT *result = (FLOAT*)malloc(SIDE * SIDE * dsize);
	memset(result, 0, SIDE * SIDE * dsize);

	FLOAT *sync= (FLOAT*)malloc(SIDE * SIDE * SIDE * dsize);
	memset(sync, 0, SIDE * SIDE * SIDE * dsize);

	FLOAT *resultd, *catd, *zerod, *syncd;
	hipMalloc((void**)&resultd, SIDE * SIDE * dsize);
	hipMalloc((void**)&catd, SIDE * SIDE * dsize);
	hipMalloc((void**)&zerod, SIDE * SIDE * dsize);
	hipMalloc((void**)&syncd, SIDE * SIDE * SIDE * dsize);

	hipMemcpy(zerod, result, SIDE * SIDE * dsize, hipMemcpyHostToDevice);
	hipMemcpy(resultd, result, SIDE * SIDE * dsize, hipMemcpyHostToDevice);
	//hipMemcpy(catd, cat, SIDE * SIDE * dsize, hipMemcpyHostToDevice);
	
	dim3 blocks(SIDE, SIDE);
	dim3 threads(SIDE);
	
	//Vz3<<<blocks,threads>>>(0, 0, -100000000, 100000000, -100000000, 100000000, 1, 0, 0, result);
	//V<<<1,1>>>(result);
	for (int i = 0; i < SIDE; i++)
	{
		hipMemset(syncd, 0, SIDE * SIDE * SIDE * dsize);
		Calculate<<<blocks,threads>>>(i, 10000, 11000, 4, 4, zerod, catd, resultd, syncd);
	}
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	printf("Error: %s\n", hipGetErrorString(error));
	
	hipMemcpy(result, resultd, SIDE * SIDE * dsize, hipMemcpyDeviceToHost);
	hipFree( resultd );
	printf("%f\n", result[(SIDE / 2) * SIDE + SIDE / 2]);
	
	return 0;
}
