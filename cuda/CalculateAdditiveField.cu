#include <stdio.h>
#include "CalculateAdditiveField.cuh"
#include "../global.h"
#include "gpu_global.h"

int CalculateAdditiveField(CUDA_FLOAT* result, int nCol, int nRow, int firstRowToCalculate, int rowsToCalculateCount, AbstractDeviceContext** deviceContexts, std::vector<unsigned char> devices_list)
{
	int returnCode = 1;

	int deviceCount = devices_list.size();

	if (deviceCount == 0)
	{
		fprintf(stderr, "No CUDA devices found\n");
		return 0;
	}
	
	memset(result, 0, nCol * nRow * dsize);

	dim3 blocks(nCol, nRow);
	dim3 threads(THREADS_COUNT);

	int pos = firstRowToCalculate * nCol;
	int maximumPos = (firstRowToCalculate + rowsToCalculateCount) * nCol;
	int currentDevice = 0;
	while (pos < maximumPos)
	{
		hipSetDevice(devices_list[currentDevice]);
		deviceContexts[currentDevice]->RunCalculation(blocks, threads, pos, maximumPos);
		
		pos += THREADS_COUNT;
		currentDevice = (currentDevice + 1) % deviceCount;
	}

	CUDA_FLOAT* result_tmp;
	hipHostAlloc((void**)&result_tmp, nCol * nRow * dsize, hipHostMallocDefault);
	
	for (int dev = 0; dev < deviceCount; dev++)
	{
		hipSetDevice(devices_list[dev]);
		hipDeviceSynchronize();
		hipError_t error = hipGetLastError();
		if (error != hipSuccess)
		{
			printf("Error: %s\n", hipGetErrorString(error));
			returnCode = 0;
		}
	
		deviceContexts[dev]->GetResult(result_tmp);

		// Accumulate result. Works for additive fields only!
		for (int i = 0; i < nCol * nRow; i++)
			result[i] += result_tmp[i];
		
		delete deviceContexts[dev];
	}

	return returnCode;
}
