#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""

#include "../global.h"

#define THREADS_COUNT 128

__device__
FLOAT Vz1(FLOAT x, FLOAT y, FLOAT xi, FLOAT nu, FLOAT z1, FLOAT z2, FLOAT H)
{
	FLOAT x_dif = (xi - x);
	FLOAT y_dif = (nu - y);
	FLOAT z_dif2 = (z2 - H);
	FLOAT z_dif1 = (z1 - H);

	FLOAT R1 = sqrt(x_dif * x_dif + y_dif * y_dif + z_dif1 * z_dif1);
	FLOAT R2 = sqrt(x_dif * x_dif + y_dif * y_dif + z_dif2 * z_dif2);

	return 
		(nu == y ? 0 : y_dif * log((x_dif + R2) / (x_dif + R1))) + 
		(xi == x ? 0 : x_dif * log((y_dif + R2) / (y_dif + R1))) -

		((z_dif2 == 0 ? 0 : z_dif2 * atan(x_dif * y_dif / (z_dif2 * R2))) -
		(z_dif1 == 0 ? 0 : z_dif1 * atan(x_dif * y_dif / (z_dif1 * R1))));
}

__device__
FLOAT Vz2(FLOAT x, FLOAT y, FLOAT xi, FLOAT y1, FLOAT y2, FLOAT z1, FLOAT z2, FLOAT H)
{
	return Vz1(x, y, xi, y2, z1, z2, H) - Vz1(x, y, xi, y1, z1, z2, H);
}

__device__
FLOAT Vz3(FLOAT x, FLOAT y, FLOAT x1, FLOAT x2, FLOAT y1, FLOAT y2, FLOAT z1, FLOAT z2, FLOAT H)
{
	return Vz2(x, y, x2, y1, y2, z1, z2, H) - Vz2(x, y, x1, y1, y2, z1, z2, H);
}

__global__
void Calculate(int first_block_pos, int nCol, FLOAT xLL, FLOAT yLL, FLOAT xStep, FLOAT yStep, FLOAT* top, FLOAT* bottom, FLOAT* result)
{
	__shared__ FLOAT sync[THREADS_COUNT];
	
	int pos_grid = (first_block_pos + threadIdx.x);
	
	int xPos = pos_grid % nCol;
	int yPos = pos_grid / nCol;
	
	FLOAT x = xLL + xStep * blockIdx.x;
	FLOAT y = yLL + yStep * blockIdx.y;
	
	FLOAT x1 = xLL + xStep * xPos;
	FLOAT x2 = x1 + xStep;
	
	FLOAT y1 = yLL + yStep * yPos;
	FLOAT y2 = y1 + yStep;

	FLOAT t = top[pos_grid];
	FLOAT b = bottom[pos_grid];
	
	int pos_result = blockIdx.x + blockIdx.y * nCol;
	
	FLOAT r = Vz3(x, y, x1, x2, y1, y2, t, b, 0);
	
	sync[threadIdx.x] = r;
	FLOAT res = result[pos_result];
	__syncthreads();
	if (threadIdx.x)
		return;

	for (int i = 0; i < THREADS_COUNT; i++)
		res += sync[i];
	result[pos_result] = res;
}

int CalculateVz(FLOAT* top, FLOAT* bottom, FLOAT* result, int nCol, int nRow, int firstRowToCalculate, int rowsToCalculateCount, FLOAT xLL, FLOAT yLL, FLOAT xSize, FLOAT ySize)
{
	int returnCode = 1;
	
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	printf("Found %d CUDA devices, ", deviceCount);
	// We need to get so many devices as we can to split data to equal-sized portions
	while (nRow % deviceCount != 0)
		deviceCount--;
	printf("using %d of them\n", deviceCount);
	if (deviceCount == 0)
		return 0;
	
	memset(result, 0, nCol * nRow * dsize);
	
	FLOAT *resultd[deviceCount], *bottomd[deviceCount], *topd[deviceCount];
	for (int dev = 0; dev < deviceCount; dev++)
	{
		hipSetDevice(dev);
		hipMalloc((void**)&resultd[dev], nCol * nRow * dsize);
		hipMalloc((void**)&bottomd[dev], nCol * nRow * dsize);
		hipMalloc((void**)&topd[dev], nCol * nRow * dsize);

		hipMemcpy(topd[dev], top, nCol * nRow * dsize, hipMemcpyHostToDevice);
		hipMemcpy(bottomd[dev], bottom, nCol * nRow * dsize, hipMemcpyHostToDevice);
		hipMemcpy(resultd[dev], result, nCol * nRow * dsize, hipMemcpyHostToDevice);
	}

	dim3 blocks(nCol, nRow);
	dim3 threads(THREADS_COUNT);

	for (int pos = firstRowToCalculate * nCol; pos < (firstRowToCalculate + rowsToCalculateCount) * nCol;)
	{
		for (int dev = 0; dev < deviceCount; dev++, pos += THREADS_COUNT)
		{
			hipSetDevice(dev);
			Calculate<<<blocks,threads>>>(pos, nCol, xLL, yLL, xSize, ySize, topd[dev], bottomd[dev], resultd[dev]);
		}
	}

	
	FLOAT* result_tmp;
	hipHostAlloc((void**)&result_tmp, nCol * nRow * dsize, hipHostMallocDefault);
	
	for (int dev = 0; dev < deviceCount; dev++)
	{
		hipSetDevice(dev);
		hipDeviceSynchronize();
		hipError_t error = hipGetLastError();
		if (error != hipSuccess)
		{
			printf("Error: %s\n", hipGetErrorString(error));
			returnCode = 0;
		}
	
		hipMemcpy(result_tmp, resultd[dev], nCol * nRow * dsize, hipMemcpyDeviceToHost);
		for (int i = 0; i < nCol * nRow; i++)
			result[i] += result_tmp[i];
		hipFree(resultd[dev]);
		hipFree(topd[dev]);
		hipFree(bottomd[dev]);
	}
	return returnCode;
}
