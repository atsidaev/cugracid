#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""

#include "../global.h"

#define LINES_PER_BLOCK 1

__device__
FLOAT Vz1(FLOAT x, FLOAT y, FLOAT xi, FLOAT nu, FLOAT z1, FLOAT z2, FLOAT H)
{
	FLOAT x_dif = (xi - x);
	FLOAT y_dif = (nu - y);
	FLOAT z_dif2 = (z2 - H);
	FLOAT z_dif1 = (z1 - H);

	FLOAT R1 = sqrt(x_dif * x_dif + y_dif * y_dif + z_dif1 * z_dif1);
	FLOAT R2 = sqrt(x_dif * x_dif + y_dif * y_dif + z_dif2 * z_dif2);

	return 
		(nu == y ? 0 : y_dif * log((x_dif + R2) / (x_dif + R1))) + 
		(xi == x ? 0 : x_dif * log((y_dif + R2) / (y_dif + R1))) -

		((z_dif2 == 0 ? 0 : z_dif2 * atan(x_dif * y_dif / (z_dif2 * R2))) -
		(z_dif1 == 0 ? 0 : z_dif1 * atan(x_dif * y_dif / (z_dif1 * R1))));
}

__device__
FLOAT Vz2(FLOAT x, FLOAT y, FLOAT xi, FLOAT y1, FLOAT y2, FLOAT z1, FLOAT z2, FLOAT H)
{
	return Vz1(x, y, xi, y2, z1, z2, H) - Vz1(x, y, xi, y1, z1, z2, H);
}

__device__
FLOAT Vz3(FLOAT x, FLOAT y, FLOAT x1, FLOAT x2, FLOAT y1, FLOAT y2, FLOAT z1, FLOAT z2, FLOAT H)
{
	return Vz2(x, y, x2, y1, y2, z1, z2, H) - Vz2(x, y, x1, y1, y2, z1, z2, H);
}

__global__
void Calculate(int yLine, FLOAT xLL, FLOAT yLL, FLOAT xStep, FLOAT yStep, FLOAT* top, FLOAT* bottom, FLOAT* result, FLOAT* sync)
{
	yLine += threadIdx.y;

	FLOAT x = xLL + xStep * blockIdx.x;
	FLOAT y = yLL + yStep * blockIdx.y;
	
	FLOAT x1 = xLL + xStep * threadIdx.x;
	FLOAT x2 = x1 + xStep;
	
	FLOAT y1 = yLL + yStep * yLine;
	FLOAT y2 = y1 + yStep;
	
	int pos_grid = threadIdx.x + yLine * SIDE;
	FLOAT t = 40.326; //top[pos_grid];
	FLOAT b = bottom[pos_grid];
	
	int pos_result = blockIdx.x + blockIdx.y * SIDE;
	
	FLOAT r = Vz3(x, y, x1, x2, y1, y2, t, b, 0);
	
	//atomicAdd(&result[pos_result], r);
	//result[pos_result] += r;
	sync[pos_result * LINES_PER_BLOCK * SIDE + threadIdx.y*SIDE + threadIdx.x] = r;
	FLOAT res = result[pos_result];
	__syncthreads();
	if (threadIdx.x)
		return;

	for (int i = 0, p = pos_result * LINES_PER_BLOCK * SIDE; i < LINES_PER_BLOCK * SIDE; i++, p++)
		res += sync[p];
	result[pos_result] = res;
	// atomicAdd(&result[pos_result], r);
}

extern "C" { 

int CalculateVz(FLOAT* top, FLOAT* bottom, FLOAT* result)
{
	int returnCode = 1;
	FLOAT *sync= (FLOAT*)malloc(LINES_PER_BLOCK * SIDE * SIDE * SIDE * dsize);
	memset(sync, 0, LINES_PER_BLOCK * SIDE * SIDE * SIDE * dsize);

	FLOAT *resultd, *bottomd, *topd, *syncd;
	hipMalloc((void**)&resultd, SIDE * SIDE * dsize);
	hipMalloc((void**)&bottomd, SIDE * SIDE * dsize);
	hipMalloc((void**)&topd, SIDE * SIDE * dsize);
	hipMalloc((void**)&syncd, LINES_PER_BLOCK * SIDE * SIDE * SIDE * dsize);

	hipMemcpy(topd, top, SIDE * SIDE * dsize, hipMemcpyHostToDevice);
	hipMemcpy(bottomd, bottom, SIDE * SIDE * dsize, hipMemcpyHostToDevice);	
	hipMemcpy(resultd, result, SIDE * SIDE * dsize, hipMemcpyHostToDevice);
	
	dim3 blocks(SIDE, SIDE);
	dim3 threads(SIDE, LINES_PER_BLOCK);
	
	for (int i = 0; i < SIDE; i+=LINES_PER_BLOCK)
	{
		hipMemset(syncd, 0, LINES_PER_BLOCK * SIDE * SIDE * SIDE * dsize);
		Calculate<<<blocks,threads>>>(i, 10017.376448317, 6395.193574, 3.0982365948353, 4.1303591058824, topd, bottomd, resultd, syncd);
	}
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("Error: %s\n", hipGetErrorString(error));
		returnCode = 0;
	}
	
	hipMemcpy(result, resultd, SIDE * SIDE * dsize, hipMemcpyDeviceToHost);
	hipFree(resultd);
	hipFree(topd);
	hipFree(bottomd);
	hipFree(syncd);
	
	return returnCode;
}

}