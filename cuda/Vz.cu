#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""

#ifdef _WIN32
#include <memory.h>
#endif

#include "../global.h"

#define THREADS_COUNT 128

__device__
FLOAT Vz1(FLOAT x, FLOAT y, FLOAT xi, FLOAT nu, FLOAT z1, FLOAT z2, FLOAT H)
{
	FLOAT x_dif = (xi - x);
	FLOAT y_dif = (nu - y);
	FLOAT z_dif2 = (z2 - H);
	FLOAT z_dif1 = (z1 - H);

	FLOAT R1 = sqrt(x_dif * x_dif + y_dif * y_dif + z_dif1 * z_dif1);
	FLOAT R2 = sqrt(x_dif * x_dif + y_dif * y_dif + z_dif2 * z_dif2);

	return 
		-((nu == y ? 0 : y_dif * log((x_dif + R2) / (x_dif + R1))) + 
		  (xi == x ? 0 : x_dif * log((y_dif + R2) / (y_dif + R1))) -

		((z_dif2 == 0 ? 0 : z_dif2 * atan(x_dif * y_dif / (z_dif2 * R2))) -
		(z_dif1 == 0 ? 0 : z_dif1 * atan(x_dif * y_dif / (z_dif1 * R1)))));
}

__device__
FLOAT Vz2(FLOAT x, FLOAT y, FLOAT xi, FLOAT y1, FLOAT y2, FLOAT z1, FLOAT z2, FLOAT H)
{
	return Vz1(x, y, xi, y2, z1, z2, H) - Vz1(x, y, xi, y1, z1, z2, H);
}

__device__
FLOAT Vz3(FLOAT x, FLOAT y, FLOAT x1, FLOAT x2, FLOAT y1, FLOAT y2, FLOAT z1, FLOAT z2, FLOAT H)
{
	return Vz2(x, y, x2, y1, y2, z1, z2, H) - Vz2(x, y, x1, y1, y2, z1, z2, H);
}

__global__
void Calculate(int first_block_pos, int maximumPos, int nCol, FLOAT xLL, FLOAT yLL, FLOAT xStep, FLOAT yStep, FLOAT* top, FLOAT* bottom, FLOAT* result)
{
	__shared__ FLOAT sync[THREADS_COUNT];
	
	int pos_grid = (first_block_pos + threadIdx.x);
	if (pos_grid >= maximumPos)
	{
		// if we are outside of data which should be calculated by our block
		// then we need to skip this thread to avoid double calculation of the nodes
		sync[threadIdx.x] = 0;
		return;
	}

	int xPos = pos_grid % nCol;
	int yPos = pos_grid / nCol;
	
	FLOAT x = xLL + xStep * blockIdx.x + xStep / 2;
	FLOAT y = yLL + yStep * blockIdx.y + yStep / 2;
	
	FLOAT x1 = xLL + xStep * xPos;
	FLOAT x2 = x1 + xStep;
	
	FLOAT y1 = yLL + yStep * yPos;
	FLOAT y2 = y1 + yStep;

	FLOAT t = top[pos_grid];
	FLOAT b = bottom[pos_grid];
	
	int pos_result = blockIdx.x + blockIdx.y * nCol;
	
	FLOAT r = Vz3(x, y, x1, x2, y1, y2, t, b, 0);
	
	// printf("Field at (%f,%f) for (%f..%f,%f..%f,%f..%f) is %f\n", x,y,x1,x2,y1,y2,t,b,r);
	
	sync[threadIdx.x] = r;
	FLOAT res = result[pos_result];
	__syncthreads();
	if (threadIdx.x)
		return;

	for (int i = 0; i < THREADS_COUNT; i++)
		res += sync[i];
	result[pos_result] = res;
}

int CalculateVz(FLOAT* top, FLOAT* bottom, FLOAT* result, int nCol, int nRow, int firstRowToCalculate, int rowsToCalculateCount, FLOAT xLL, FLOAT yLL, FLOAT xSize, FLOAT ySize)
{
	int returnCode = 1;
	
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	printf("Found %d CUDA devices, ", deviceCount);
	if (deviceCount == 0)
		return 0;
	
	memset(result, 0, nCol * nRow * dsize);
	
	FLOAT **resultd, **bottomd, **topd;
	resultd = new FLOAT*[deviceCount];
	bottomd = new FLOAT*[deviceCount];
	topd = new FLOAT*[deviceCount];

	// Setup inbound and outbound arrays for all CUDA devices
	for (int dev = 0; dev < deviceCount; dev++)
	{
		hipSetDevice(dev);
		hipMalloc((void**)&resultd[dev], nCol * nRow * dsize);
		hipMalloc((void**)&bottomd[dev], nCol * nRow * dsize);
		hipMalloc((void**)&topd[dev], nCol * nRow * dsize);

		hipMemcpy(topd[dev], top, nCol * nRow * dsize, hipMemcpyHostToDevice);
		hipMemcpy(bottomd[dev], bottom, nCol * nRow * dsize, hipMemcpyHostToDevice);
		hipMemcpy(resultd[dev], result, nCol * nRow * dsize, hipMemcpyHostToDevice);
	}

	dim3 blocks(nCol, nRow);
	dim3 threads(THREADS_COUNT);

	int pos = firstRowToCalculate * nCol;
	int maximumPos = (firstRowToCalculate + rowsToCalculateCount) * nCol;
	int currentDevice = 0;
	while (pos < maximumPos)
	{
		hipSetDevice(currentDevice);
		Calculate<<<blocks,threads>>>(pos, maximumPos, nCol, xLL, yLL, xSize, ySize, topd[currentDevice], bottomd[currentDevice], resultd[currentDevice]);
		pos += THREADS_COUNT;
		currentDevice = (currentDevice + 1) % deviceCount;
	}

	FLOAT* result_tmp;
	hipHostAlloc((void**)&result_tmp, nCol * nRow * dsize, hipHostMallocDefault);
	
	for (int dev = 0; dev < deviceCount; dev++)
	{
		hipSetDevice(dev);
		hipDeviceSynchronize();
		hipError_t error = hipGetLastError();
		if (error != hipSuccess)
		{
			printf("Error: %s\n", hipGetErrorString(error));
			returnCode = 0;
		}
	
		hipMemcpy(result_tmp, resultd[dev], nCol * nRow * dsize, hipMemcpyDeviceToHost);
		for (int i = 0; i < nCol * nRow; i++)
			result[i] += result_tmp[i];
		hipFree(resultd[dev]);
		hipFree(topd[dev]);
		hipFree(bottomd[dev]);
	}

	delete[] resultd;
	delete[] topd;
	delete[] bottomd;

	return returnCode;
}
